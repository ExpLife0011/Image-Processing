#include "hip/hip_runtime.h"
#include <cstdlib>
#include <algorithm>

#include "Utils/utils.hpp"

#include "feature_tracking_gpu.cuh"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include ""

__device__ struct d_PointData {
	d_Point location;
	float corner_response;
	float signature[49];
};

__device__ __forceinline__ uint d_idx_1d(uint x, uint y, uint width) {
	return (width * y) + x;
}

__device__ __forceinline__ float d_get_template_average(float *signature) {
	float template_average = 0.0f;
	for(uchar i=0; i<49; ++i) {
		template_average += signature[i];
	}
	return template_average /= 49.0f;
}

__device__ float d_get_window_average(
	uint x, uint y,
	uint num_cols, uint num_rows,
	float * __restrict normalized_input_image)
{
	float window_average = 0.0f;
	for(int window_offset_y=-3; window_offset_y<=3; ++window_offset_y) {
		for(int window_offset_x=-3; window_offset_x<=3; ++window_offset_x) {
			int window_x = x + window_offset_x;
			int window_y = y + window_offset_y;
			window_x = window_x >= num_cols ? num_cols-1 : window_x < 0 ? 0 : window_y;
			window_y = window_y >= num_rows ? num_rows-1 : window_y < 0 ? 0 : window_y;
			window_average += normalized_input_image[d_idx_1d(window_x, window_y, num_cols)];
		}
	}
	return window_average /= 49.0f;
}

__global__ void _create_normalised_input_image(
	uchar * __restrict input_image,
	float * __restrict uchar_normalize_table,
	float * __restrict normalized_input_image,
	uint num_cols,
	uint num_rows)
{
	const uint2 thread_2D_pos = make_uint2(
		blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y
	);

	const uint thread_1D_pos = d_idx_1d(thread_2D_pos.x, thread_2D_pos.y, num_cols);

	if(thread_2D_pos.x >= num_cols || thread_2D_pos.y >= num_rows) {
		return;
	}

	normalized_input_image[thread_1D_pos] = uchar_normalize_table[input_image[thread_1D_pos]];
}

__global__ void _calc_gradients(
	uchar * __restrict input_image,
	short * __restrict gradient_x2,
	short * __restrict gradient_y2,
	short * __restrict gradient_xy,
	char * __restrict sobel_x,
	char * __restrict sobel_y,
	uint num_cols,
	uint gradient_cols,
	uint gradient_rows)
{
	const uint2 thread_2D_pos = make_uint2(
		blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y
	);

	const uint thread_1D_pos = d_idx_1d(thread_2D_pos.x, thread_2D_pos.y, gradient_cols);

	if(thread_2D_pos.x >= gradient_cols || thread_2D_pos.y >= gradient_rows) {
		return;
	}

	const uint x = thread_2D_pos.x + 1;
	const uint y = thread_2D_pos.y + 1;

	const short gradient_x = (
		(sobel_x[0] * input_image[d_idx_1d(x-1, y-1, num_cols)]) +
		(sobel_x[1] * input_image[d_idx_1d(x+0, y-1, num_cols)]) +
		(sobel_x[2] * input_image[d_idx_1d(x+1, y-1, num_cols)]) +
		(sobel_x[3] * input_image[d_idx_1d(x-1, y+0, num_cols)]) +
		(sobel_x[4] * input_image[d_idx_1d(x+0, y+0, num_cols)]) +
		(sobel_x[5] * input_image[d_idx_1d(x+1, y+0, num_cols)]) +
		(sobel_x[6] * input_image[d_idx_1d(x-1, y+1, num_cols)]) +
		(sobel_x[7] * input_image[d_idx_1d(x+0, y+1, num_cols)]) +
		(sobel_x[8] * input_image[d_idx_1d(x+1, y+1, num_cols)])
	);

	const short gradient_y = (
		(sobel_y[0] * input_image[d_idx_1d(x-1, y-1, num_cols)]) +
		(sobel_y[1] * input_image[d_idx_1d(x+0, y-1, num_cols)]) +
		(sobel_y[2] * input_image[d_idx_1d(x+1, y-1, num_cols)]) +
		(sobel_y[3] * input_image[d_idx_1d(x-1, y+0, num_cols)]) +
		(sobel_y[4] * input_image[d_idx_1d(x+0, y+0, num_cols)]) +
		(sobel_y[5] * input_image[d_idx_1d(x+1, y+0, num_cols)]) +
		(sobel_y[6] * input_image[d_idx_1d(x-1, y+1, num_cols)]) +
		(sobel_y[7] * input_image[d_idx_1d(x+0, y+1, num_cols)]) +
		(sobel_y[8] * input_image[d_idx_1d(x+1, y+1, num_cols)])
	);

	gradient_x2[thread_1D_pos] = gradient_x * gradient_x;
	gradient_y2[thread_1D_pos] = gradient_y * gradient_y;
	gradient_xy[thread_1D_pos] = gradient_x * gradient_y;
}

__global__ void _blur_gradients(
	short * __restrict gradient_image,
	float * __restrict blur_gradient_image,
	float *__restrict gaussian_matrix,
	char filter_range,
	uint gradient_cols,
	uint blur_gradient_cols,
	uint blur_gradient_rows)
{
	const uint2 thread_2D_pos = make_uint2(
		blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y
	);

	const uint thread_1D_pos = d_idx_1d(thread_2D_pos.x, thread_2D_pos.y, blur_gradient_cols);

	if(thread_2D_pos.x >= blur_gradient_cols || thread_2D_pos.y >= blur_gradient_rows) {
		return;
	}

	float total = 0.0f;
	for(uint y=thread_2D_pos.y-filter_range, gauss_idx=0; y<thread_2D_pos.y+filter_range; ++y, ++gauss_idx) {
		for(uint x=thread_2D_pos.x-filter_range; x<thread_2D_pos.x+filter_range; ++x, ++gauss_idx) {
			total += gaussian_matrix[gauss_idx] * gradient_image[d_idx_1d(x+filter_range, y+filter_range, gradient_cols)];
		}
	}
	blur_gradient_image[d_idx_1d(thread_2D_pos.x, thread_2D_pos.y, blur_gradient_cols)] = total;
}

__global__ void _calc_harris_response(
	float * __restrict blur_gradient_x2,
	float * __restrict blur_gradient_y2,
	float * __restrict blur_gradient_xy,
	float * __restrict harris_response,
	float sensitivity,
	uint blur_gradient_cols,
	uint blur_gradient_rows)
{
	const uint2 thread_2D_pos = make_uint2(
		blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y
	);

	const uint thread_1D_pos = d_idx_1d(thread_2D_pos.x, thread_2D_pos.y, blur_gradient_cols);

	if(thread_2D_pos.x >= blur_gradient_cols || thread_2D_pos.y >= blur_gradient_rows) {
		return;
	}

	float gx2 = blur_gradient_x2[thread_1D_pos];
	float gy2 = blur_gradient_y2[thread_1D_pos];
	float gxy = blur_gradient_xy[thread_1D_pos];

	float det = (gx2 * gy2) - (gxy * gxy);
	float trace = gx2 + gy2;

	harris_response[thread_1D_pos] = det - (sensitivity * (trace * trace));
}

__global__ void _non_maximum_suppression(
	float * __restrict harris_response,
	bool * __restrict maxima_suppression,
	int maxima_suppression_range,
	float threshhold,
	uint harris_response_cols,
	uint harris_response_rows,
	uint * __restrict points_after_suppression)
{
	const uint2 thread_2D_pos = make_uint2(
		blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y
	);

	const uint thread_1D_pos = d_idx_1d(thread_2D_pos.x, thread_2D_pos.y, harris_response_cols);

	if(thread_2D_pos.x >= harris_response_cols || thread_2D_pos.y >= harris_response_rows) {
		return;
	}

	float current_response = harris_response[thread_1D_pos];

	for(char window_offset_y=-maxima_suppression_range; window_offset_y<=maxima_suppression_range; ++window_offset_y) {
		for(char window_offset_x=-maxima_suppression_range; window_offset_x<=maxima_suppression_range; ++window_offset_x) {
			int window_x = thread_2D_pos.x + window_offset_x;
			int window_y = thread_2D_pos.y + window_offset_y;
			if(window_x < 0 || window_x >= harris_response_cols || window_y < 0 || window_y >= harris_response_rows) {
				break;
			}
			if(window_offset_x == 0 && window_offset_y == 0) {
				if(current_response < threshhold) {
					maxima_suppression[thread_1D_pos] = false;
				}
				break;
			}

			uint window_idx = d_idx_1d(window_x, window_y, harris_response_cols);
			float window_point_response = harris_response[window_idx];

			if(current_response > window_point_response) {
				maxima_suppression[window_idx] = false;
			}
		}
	}

	__syncthreads();

	if(maxima_suppression[thread_1D_pos] == true) {
		atomicAdd(points_after_suppression, 1);
	}
}

__global__ void _fill_points(
	float * __restrict harris_response,
	bool * __restrict maxima_suppression,
	d_PointData * __restrict points,
	char filter_range,
	uint harris_response_cols,
	uint harris_response_rows,
	uint num_cols,
	uint num_rows,
	float * __restrict normalized_input_image,
	uint * __restrict points_after_suppression)
{
	const uint2 thread_2D_pos = make_uint2(
		blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y
	);

	const uint thread_1D_pos = d_idx_1d(thread_2D_pos.x, thread_2D_pos.y, harris_response_cols);

	if(thread_2D_pos.x >= harris_response_cols || thread_2D_pos.y >= harris_response_rows) {
		return;
	}

	if(maxima_suppression[thread_1D_pos]) {
		const uint write_idx = atomicAdd(points_after_suppression, 1);

		d_Point point;
		point.x = thread_2D_pos.x + 1 + filter_range;
		point.y = thread_2D_pos.y + 1 + filter_range;

		d_PointData point_data;
		point_data.corner_response = harris_response[thread_1D_pos];
		point_data.location = point;

		for(char window_offset_y=-3, template_y=0; window_offset_y<=3; ++window_offset_y, ++template_y) {
			for(char window_offset_x=-3, template_x=0; window_offset_x<=3; ++window_offset_x, ++template_x) {
				int window_x = point.x + window_offset_x;
				int window_y = point.y + window_offset_y;
				window_x = window_x >= num_cols ? num_cols-1 : window_x < 0 ? 0 : window_x;
				window_y = window_y >= num_rows ? num_rows-1 : window_y < 0 ? 0 : window_y;
				point_data.signature[(template_y * 7) + template_x] = normalized_input_image[d_idx_1d(window_x, window_y, num_cols)];
			}
		}

		points[write_idx] = point_data;
	}
}

__device__ float calc_correlation(
	int search_area_x,
	int search_area_y,
	float * __restrict signature,
	float * __restrict normalized_input_image,
	uint num_cols,
	uint num_rows)
{
	const float template_average = d_get_template_average(signature);

	float ixy = 0.0f;
	float ix2 = 0.0f;
	float iy2 = 0.0f;
	for(char window_offset_y=-3, template_y=0; window_offset_y<=3; ++window_offset_y, ++template_y) {
		for(char window_offset_x=-3, template_x=0; window_offset_x<=3; ++window_offset_x, ++template_x) {
			int window_x = search_area_x + window_offset_x;
			int window_y = search_area_y + window_offset_y;
			window_x = window_x >= num_cols ? num_cols-1 : window_x < 0 ? 0 : window_x;
			window_y = window_y >= num_rows ? num_rows-1 : window_y < 0 ? 0 : window_y;

			float window_average = d_get_window_average(window_x, window_y, num_cols, num_rows, normalized_input_image);

			float pixel_value = normalized_input_image[d_idx_1d(window_x, window_y, num_cols)];
			float template_value = signature[(template_y * 7) + template_x];

			float ix = pixel_value - window_average;
			float iy = template_value - template_average;

			ixy += ix * iy;
			ix2 += ix * ix;
			iy2 += iy * iy;
		}
	}

	return ixy / sqrt(ix2 * iy2);
}

/* Evaluate correlation value of each pixel in an area around the current tracked feature */
__global__ void _calc_correlation_values(
	d_HarrisPoint * __restrict tracked_features,
	float * __restrict normalized_input_image,
	d_Correlation * __restrict correlation_map,
	d_Correlation * __restrict correlation_map_new_template,
	uint template_update_frames,
	uint num_cols,
	uint num_rows)
{
	const uint2 thread_2D_pos = make_uint2(
		threadIdx.x,
		threadIdx.y
	);
	const uint thread_1D_pos = d_idx_1d(thread_2D_pos.x, thread_2D_pos.y, 7);
	const uint feature_idx = blockIdx.x;
	const uint correlation_idx = (feature_idx * 49) + thread_1D_pos;

	d_HarrisPoint *feature = &tracked_features[feature_idx];

	int search_area_offset_x = thread_2D_pos.x - 3;
	int search_area_offset_y = thread_2D_pos.y - 3;
	
	/* Add offset to current tracked feature to get X and Y coordinates of point
	in the search area currently being evaluated for correlation */
	int search_area_x = feature->locations[feature->location_idx].x + search_area_offset_x;
	int search_area_y = feature->locations[feature->location_idx].y + search_area_offset_y;

	correlation_map[correlation_idx].location.x = search_area_x;
	correlation_map[correlation_idx].location.y = search_area_y;
	correlation_map_new_template[correlation_idx].location.x = search_area_x;
	correlation_map_new_template[correlation_idx].location.y = search_area_y;

	/* Return if this point in the window is outside the image */
	if(search_area_x>=num_cols || search_area_x<0 || search_area_y>=num_rows || search_area_y<0) {
		correlation_map[correlation_idx].correlation = -100000;
		correlation_map_new_template[correlation_idx].correlation = -100000;
		return;
	}

	/* Calculate and store correlation value for the current search area pixel */
	correlation_map[correlation_idx].correlation = calc_correlation(
		search_area_x, search_area_y,
		feature->signature,
		normalized_input_image,
		num_cols, num_rows
	);
	if((feature->track_frames + 1) % (template_update_frames * 2) == 0) {
		correlation_map_new_template[correlation_idx].correlation = calc_correlation(
			search_area_x, search_area_y,
			feature->new_signature,
			normalized_input_image,
			num_cols, num_rows
		);
	}
}

__device__ d_Correlation get_max_correlation(d_Correlation *correlation_map) {
	d_Correlation max_correlation;
	max_correlation.correlation = -100000;
	for(uchar i=0; i<49; ++i) {
		if(correlation_map[i].correlation > max_correlation.correlation) {
			max_correlation = correlation_map[i];
		}
	}
	return max_correlation;
}

__device__ __forceinline__ float d_distance(d_Point p1, d_Point p2) {
	const float diff_x = (long)p1.x - (long)p2.x;
	const float diff_y = (long)p1.y - (long)p2.y;
	return std::sqrt((diff_x * diff_x) + (diff_y * diff_y));
}

__global__ void _update_tracked_features(
	d_HarrisPoint * __restrict tracked_features,
	float * __restrict normalized_input_image,
	d_Correlation * __restrict correlation_map,
	d_Correlation * __restrict correlation_map_new_template,
	uint num_tracked,
	uint num_cols,
	uint num_rows,
	float correlation_threshhold,
	uint template_update_frames,
	float template_update_distance_threshhold)
{
	const uint idx = blockDim.x * blockIdx.x + threadIdx.x;

	if(idx >= num_tracked) {
		return;
	}

	d_HarrisPoint *feature = &tracked_features[idx];
	const uint correlation_idx = idx * 49;

	bool track_success;
	d_Point new_location;
	d_Correlation max_correlation = get_max_correlation(&correlation_map[correlation_idx]);

	if((feature->track_frames + 1) % (template_update_frames * 2) == 0) {
		d_Correlation max_correlation_new_template = get_max_correlation(&correlation_map_new_template[correlation_idx]);
		if(max_correlation_new_template.correlation >= correlation_threshhold && d_distance(max_correlation.location, max_correlation_new_template.location) < template_update_distance_threshhold) {
			new_location = max_correlation_new_template.location;
			track_success = true;
		} else {
			track_success = false;
		}
	} else {
		new_location = max_correlation.location;
		track_success = max_correlation.correlation >= correlation_threshhold;
	}

	if(track_success) {
		++feature->track_frames;

		feature->location_idx = (feature->location_idx + 1) % MAX_TRACKED_POINT_LOCATIONS;
		feature->locations[feature->location_idx] = new_location;

		/* If we have tracked this point for enough frames to trigger template updating */
		if(feature->track_frames % template_update_frames == 0 && (feature->track_frames + 1) % (template_update_frames * 2) != 0) {
			/* Update the tracked feature's 7x7 template to that of its current location in the image */
			for(char window_offset_y=-3, template_y=0; window_offset_y<=3; ++window_offset_y, ++template_y) {
				for(char window_offset_x=-3, template_x=0; window_offset_x<=3; ++window_offset_x, ++template_x) {
					int window_x = new_location.x + window_offset_x;
					int window_y = new_location.y + window_offset_y;
					window_x = window_x >= num_cols ? num_cols-1 : window_x < 0 ? 0 : window_x;
					window_y = window_y >= num_rows ? num_rows-1 : window_y < 0 ? 0 : window_y;

					feature->new_signature[(template_y * 7) + template_x] = normalized_input_image[d_idx_1d(window_x, window_y, num_cols)];
				}
			}
		}
	} else {
		/* Correlation value was not above threshhold
		feature has been lost and will no longer be tracked */
		feature->tracked = false;
	}
}

struct sort_by_corner_response {
	bool operator()(PointData const &left, PointData const &right) {
		return left.corner_response > right.corner_response;
	}
};

FeatureTrackingGpu::FeatureTrackingGpu(int cuda_device, const TrackingSettings &tracking_settings) :
	FeatureTracking(),
	device(cuda_device),
	settings(tracking_settings)
{
	hipSetDevice(device);

	gradient_cols = image_width - 2;
	gradient_rows = image_height - 2;

	blur_gradient_cols = gradient_cols - (filter_range * 2);
	blur_gradient_rows = gradient_rows - (filter_range * 2);

	harris_response_cols = blur_gradient_cols;
	harris_response_rows = blur_gradient_rows;

	block_side_len = 32;
	block_size = dim3(block_side_len, block_side_len, 1);
	input_image_grid_size = dim3(image_width/block_side_len + 1, image_height/block_side_len + 1, 1);
	gradient_grid_size = dim3(gradient_cols/block_side_len + 1, gradient_rows/block_side_len + 1, 1);
	blur_gradient_grid_size = dim3(blur_gradient_cols/block_side_len + 1, blur_gradient_rows/block_side_len + 1, 1);
	harris_response_grid_size = dim3(harris_response_cols/block_side_len + 1, harris_response_rows/block_side_len + 1, 1);

	input_image_size = image_width * image_height * sizeof(uchar);

	checkCudaErrors(hipMalloc(&d_input_image, input_image_size));
	checkCudaErrors(hipMalloc(&d_normalized_input_image, image_width * image_height * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_tracked_feature_map, image_width * image_height * sizeof(bool)));
	checkCudaErrors(hipMemset(d_tracked_feature_map, false, image_width * image_height * sizeof(bool)));
	checkCudaErrors(hipMalloc(&d_sobel_x, 9 * sizeof(char)));
	checkCudaErrors(hipMalloc(&d_sobel_y, 9 * sizeof(char)));
	checkCudaErrors(hipMemcpy(d_sobel_x, sobel_x, 9 * sizeof(char), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_sobel_y, sobel_y, 9 * sizeof(char), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc(&d_gaussian_matrix, filter_width * filter_width * sizeof(float)));
	checkCudaErrors(hipMemcpy(d_gaussian_matrix, gaussian_matrix, filter_width * filter_width * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc(&d_uchar_normalize_table, 256 * sizeof(float)));
	checkCudaErrors(hipMemcpy(d_uchar_normalize_table, uchar_normalize_table, 256 * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc(&d_gradient_x2, gradient_cols * gradient_rows * sizeof(short)));
	checkCudaErrors(hipMalloc(&d_gradient_y2, gradient_cols * gradient_rows * sizeof(short)));
	checkCudaErrors(hipMalloc(&d_gradient_xy, gradient_cols * gradient_rows * sizeof(short)));
	checkCudaErrors(hipMalloc(&d_blur_gradient_x2, blur_gradient_cols * blur_gradient_rows * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_blur_gradient_y2, blur_gradient_cols * blur_gradient_rows * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_blur_gradient_xy, blur_gradient_cols * blur_gradient_rows * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_harris_response, harris_response_cols * harris_response_rows * sizeof(float)));
	checkCudaErrors(hipMalloc(&d_maxima_suppression, harris_response_cols * harris_response_rows * sizeof(bool)));
	checkCudaErrors(hipMalloc(&d_points_after_suppression, sizeof(uint)));
	checkCudaErrors(hipMalloc(&d_tracked_features, settings.max_tracked_features * sizeof(d_HarrisPoint)));
	checkCudaErrors(hipMalloc(&d_correlation_map, settings.max_tracked_features * 49 * sizeof(d_Correlation)));
	checkCudaErrors(hipMalloc(&d_correlation_map_new_template, settings.max_tracked_features * 49 * sizeof(d_Correlation)));

	h_tracked_features = (HarrisPoint *)malloc(settings.max_tracked_features * sizeof(HarrisPoint));
	h_tracked_feature_map = (bool *)malloc(image_width * image_height * sizeof(bool));
}

FeatureTrackingGpu::~FeatureTrackingGpu() {
	checkCudaErrors(hipFree(d_input_image));
	checkCudaErrors(hipFree(d_normalized_input_image));
	checkCudaErrors(hipFree(d_tracked_feature_map));
	checkCudaErrors(hipFree(d_sobel_x));
	checkCudaErrors(hipFree(d_sobel_y));
	checkCudaErrors(hipFree(d_gaussian_matrix));
	checkCudaErrors(hipFree(d_uchar_normalize_table));
	checkCudaErrors(hipFree(d_gradient_x2));
	checkCudaErrors(hipFree(d_gradient_y2));
	checkCudaErrors(hipFree(d_gradient_xy));
	checkCudaErrors(hipFree(d_blur_gradient_x2));
	checkCudaErrors(hipFree(d_blur_gradient_y2));
	checkCudaErrors(hipFree(d_blur_gradient_xy));
	checkCudaErrors(hipFree(d_harris_response));
	checkCudaErrors(hipFree(d_maxima_suppression));
	checkCudaErrors(hipFree(d_points_after_suppression));
	checkCudaErrors(hipFree(d_tracked_features));
	checkCudaErrors(hipFree(d_correlation_map));
	checkCudaErrors(hipFree(d_correlation_map_new_template));

	free(h_tracked_features);
	free(h_tracked_feature_map);
}

void FeatureTrackingGpu::create_normalised_input_image() {
	_create_normalised_input_image<<<input_image_grid_size, block_size>>>(
		d_input_image,
		d_uchar_normalize_table,
		d_normalized_input_image,
		image_width,
		image_height
	);
}

void FeatureTrackingGpu::calc_gradients() {
	_calc_gradients<<<gradient_grid_size, block_size>>>(
		d_input_image,
		d_gradient_x2,
		d_gradient_y2,
		d_gradient_xy,
		d_sobel_x,
		d_sobel_y,
		image_width,
		gradient_cols,
		gradient_rows
	);
}

void FeatureTrackingGpu::blur_gradients() {
	_blur_gradients<<<blur_gradient_grid_size, block_size>>>(
		d_gradient_x2,
		d_blur_gradient_x2,
		d_gaussian_matrix,
		filter_range,
		gradient_cols,
		blur_gradient_cols,
		blur_gradient_rows
	);

	_blur_gradients<<<blur_gradient_grid_size, block_size>>>(
		d_gradient_y2,
		d_blur_gradient_y2,
		d_gaussian_matrix,
		filter_range,
		gradient_cols,
		blur_gradient_cols,
		blur_gradient_rows
	);

	_blur_gradients<<<blur_gradient_grid_size, block_size>>>(
		d_gradient_xy,
		d_blur_gradient_xy,
		d_gaussian_matrix,
		filter_range,
		gradient_cols,
		blur_gradient_cols,
		blur_gradient_rows
	);
}

void FeatureTrackingGpu::calc_harris_response() {
	_calc_harris_response<<<blur_gradient_grid_size, block_size>>>(
		d_blur_gradient_x2,
		d_blur_gradient_y2,
		d_blur_gradient_xy,
		d_harris_response,
		settings.sensitivity,
		blur_gradient_cols,
		blur_gradient_rows
	);
}

void FeatureTrackingGpu::get_maxima_points() {
	checkCudaErrors(hipMemset(d_maxima_suppression, 1, harris_response_cols * harris_response_rows * sizeof(bool)));
	checkCudaErrors(hipMemset(d_points_after_suppression, 0, sizeof(uint)));
	_non_maximum_suppression<<<harris_response_grid_size, block_size>>>(
		d_harris_response,
		d_maxima_suppression,
		maxima_suppression_range,
		settings.harris_response_threshhold,
		harris_response_cols,
		harris_response_rows,
		d_points_after_suppression
	);
	checkCudaErrors(hipDeviceSynchronize());

	uint num_points;
	checkCudaErrors(hipMemcpy(
		&num_points,
		d_points_after_suppression,
		sizeof(uint),
		hipMemcpyDeviceToHost
	));

	d_PointData *d_points;
	checkCudaErrors(hipMalloc(&d_points, num_points * sizeof(d_PointData)));
	checkCudaErrors(hipMemset(d_points_after_suppression, 0, sizeof(uint)));
	_fill_points<<<harris_response_grid_size, block_size>>>(
		d_harris_response,
		d_maxima_suppression,
		d_points,
		filter_range,
		harris_response_cols,
		harris_response_rows,
		image_width,
		image_height,
		d_normalized_input_image,
		d_points_after_suppression
	);
	checkCudaErrors(hipDeviceSynchronize());

	PointData *h_points = (PointData *)malloc(num_points * sizeof(PointData));
	hipMemcpy(h_points, d_points, num_points * sizeof(PointData), hipMemcpyDeviceToHost);
	checkCudaErrors(hipFree(d_points));

	std::sort(&h_points[0], &h_points[num_points-1], sort_by_corner_response());

	harris_points.clear();
#pragma loop(hint_parallel(MAX_AP_THREADS))
	for(uint i=0; i<settings.max_tracked_features && i<num_points; ++i) {
		HarrisPoint harris_point;
		harris_point.locations[0] = h_points[i].location;
		memcpy(harris_point.signature, h_points[i].signature, 49 * sizeof(float));
		++harris_point.track_frames;
		harris_point.tracked = true;
		harris_points.push_back(harris_point);
	}

	free(h_points);
}

void FeatureTrackingGpu::update_tracked_features() {
	if(image_count == 0) {
		tracked_features = harris_points;
		return;
	}

	const size_t num_tracked = tracked_features.size();

	if(num_tracked == 0) {
		return;
	}

	checkCudaErrors(hipMemcpy(d_tracked_features, &tracked_features[0], num_tracked * sizeof(HarrisPoint), hipMemcpyHostToDevice));

	_calc_correlation_values<<<dim3(num_tracked, 1, 1), dim3(7, 7, 1)>>>(
		d_tracked_features,
		d_normalized_input_image,
		d_correlation_map,
		d_correlation_map_new_template,
		settings.template_update_frames,
		image_width,
		image_height
	);
	checkCudaErrors(hipDeviceSynchronize());

	_update_tracked_features<<<1, num_tracked>>>(
		d_tracked_features,
		d_normalized_input_image,
		d_correlation_map,
		d_correlation_map_new_template,
		num_tracked,
		image_width,
		image_height,
		settings.correlation_threshhold,
		settings.template_update_frames,
		settings.template_update_distance_threshhold
	);
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipMemcpy(h_tracked_features, d_tracked_features, num_tracked * sizeof(HarrisPoint), hipMemcpyDeviceToHost));

	memset(h_tracked_feature_map, 0, image_width * image_height * sizeof(bool));
	tracked_features.clear();
#pragma loop(hint_parallel(MAX_AP_THREADS))
	for(size_t i=0; i<num_tracked; ++i) {
		if(h_tracked_features[i].tracked) {
			const uint x = h_tracked_features[i].locations[h_tracked_features[i].location_idx].x;
			const uint y = h_tracked_features[i].locations[h_tracked_features[i].location_idx].y;
			h_tracked_feature_map[idx_1d(x, y, image_width)];
			tracked_features.push_back(h_tracked_features[i]);
		}
	}

	const uint num_harris_points = harris_points.size();
#pragma loop(hint_parallel(MAX_AP_THREADS))
	for(size_t i=0; (i<num_harris_points) && (tracked_features.size()<settings.max_tracked_features); ++i) {
		const uint x = harris_points[i].locations[harris_points[i].location_idx].x;
		const uint y = harris_points[i].locations[harris_points[i].location_idx].y;
		for(char window_offset_y=-3; window_offset_y<=3; ++window_offset_y) {
			for(char window_offset_x=-3; window_offset_x<=3; ++window_offset_x) {
				int window_x = x + window_offset_x;
				int window_y = y + window_offset_y;
				window_x = window_x >= image_width ? image_width-1 : window_x < 0 ? 0 : window_x;
				window_y = window_y >= image_height ? image_height-1 : window_y < 0 ? 0 : window_y;

				if(h_tracked_feature_map[idx_1d(window_x, window_y, image_width)] == true) {
					goto NEXT_POINT;
				}
			}
		}
		tracked_features.push_back(harris_points[i]);
		h_tracked_feature_map[idx_1d(x, y, image_width)] = true;
		if(tracked_features.size() >= settings.max_tracked_features) {
			break;
		}
NEXT_POINT:;
	}
}

std::vector<HarrisPoint> FeatureTrackingGpu::feature_points(uchar *input) {
	h_input_image = input;

	checkCudaErrors(hipMemcpy(d_input_image, h_input_image, input_image_size, hipMemcpyHostToDevice));

	create_normalised_input_image();
	calc_gradients();
	checkCudaErrors(hipDeviceSynchronize());

	blur_gradients();
	checkCudaErrors(hipDeviceSynchronize());

	calc_harris_response();
	checkCudaErrors(hipDeviceSynchronize());

	get_maxima_points();
	update_tracked_features();

	++image_count;
	
	return tracked_features;
}
